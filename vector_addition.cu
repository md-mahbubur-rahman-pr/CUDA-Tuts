#include "hip/hip_runtime.h"
#define N 99999

__global__  void vector_addition (int *a, int *b, int *c, int n) {
  for (int i=0; i<n; i++) {
    c[i] = a[i] + b[i];
  }
}

int main(){
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  
  // allocate memory 
  a = (int*) malloc (sizeof(int) * N);
  b = (int*) malloc (sizeof(int) * N);
  c = (int*) malloc (sizeof(int) * N);
  
  // initialize array 
  for (int i=0; i<N; i++) {
    a[i] = 1;
    b[i] = 2;
  }
  
  // Allocate device memory
  hipMalloc ( (void**)&dev_a, sizeof(int) * N);
  hipMalloc ( (void**)&dev_b, sizeof(int) * N);
  hipMalloc ( (void**)&dev_c, sizeof(int) * N);
  
  // transfer data from host to device memory 
  hipMemcpy (dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
  
  // execute kernel 
  vector_addition<<<1,1>>>(dev_a,dev_b,dev_c,N);
  
  // transfer data back to host memory 
  hipMemcpy (c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);
  
  // display the result
  for (int i=0; i<N; i++) {
    printf("%d + %d = %d\n", ,a[i] + b[i] = c[i]);
  }
  
  // deallocate the device memory 
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  
  // deallocate host memory 
  free(a);
  free(b);
  free(c);
  
}
