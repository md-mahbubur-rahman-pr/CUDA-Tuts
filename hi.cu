
#include <hip/hip_runtime.h>

// The __global__ specifier indicates a function that runs on GPU (the device).
__global__ void cuda_say_hi() {
  printf("Hi, CUDA!\n");
}

int main() {
  // While a kernel being called, its execution configuration is provided through <<< .,. >>> syntax. 
  // In CUDA, it's called "kernel_launch".
  cuda_say_hi<<<1,1>>>();
  return 0;
}
